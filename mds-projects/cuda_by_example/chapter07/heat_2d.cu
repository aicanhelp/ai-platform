#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "book.h"
#include "image.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

// these exist on the GPU side
texture<float,2>  texConstSrc;
texture<float,2>  texIn;
texture<float,2>  texOut;

__global__ void blend_kernel( float *dst,
                              bool dstOut ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b;
    if (dstOut) {
        t = tex2D(texIn,x,y-1);
        l = tex2D(texIn,x-1,y);
        c = tex2D(texIn,x,y);
        r = tex2D(texIn,x+1,y);
        b = tex2D(texIn,x,y+1);
    } else {
        t = tex2D(texOut,x,y-1);
        l = tex2D(texOut,x-1,y);
        c = tex2D(texOut,x,y);
        r = tex2D(texOut,x+1,y);
        b = tex2D(texOut,x,y+1);
    }
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void copy_const_kernel( float *iptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex2D(texConstSrc,x,y);
    if (c != 0)
        iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *output_bitmap;
    float           *dev_inSrc;
    float           *dev_outSrc;
    float           *dev_constSrc;
    IMAGE           *bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};


// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) 
{
    hipUnbindTexture( texIn );
    hipUnbindTexture( texOut );
    hipUnbindTexture( texConstSrc );
    HANDLE_ERROR( hipFree( d->dev_inSrc ) );
    HANDLE_ERROR( hipFree( d->dev_outSrc ) );
    HANDLE_ERROR( hipFree( d->dev_constSrc ) );

    HANDLE_ERROR( hipEventDestroy( d->start ) );
    HANDLE_ERROR( hipEventDestroy( d->stop ) );
}


int main( void ) {
    DataBlock   data;
    IMAGE bitmap_image( DIM, DIM );
    data.bitmap = &bitmap_image;
    data.totalTime = 0;
    data.frames = 0;
    HANDLE_ERROR( hipEventCreate( &data.start ) );
    HANDLE_ERROR( hipEventCreate( &data.stop ) );

    int imageSize = bitmap_image.image_size();

    HANDLE_ERROR( hipMalloc( (void**)&data.output_bitmap,
                               imageSize ) );

    // assume float == 4 chars in size (ie rgba)
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_inSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_outSrc,
                              imageSize ) );
    HANDLE_ERROR( hipMalloc( (void**)&data.dev_constSrc,
                              imageSize ) );

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    HANDLE_ERROR( hipBindTexture2D( NULL, texConstSrc,
                                   data.dev_constSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texIn,
                                   data.dev_inSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    HANDLE_ERROR( hipBindTexture2D( NULL, texOut,
                                   data.dev_outSrc,
                                   desc, DIM, DIM,
                                   sizeof(float) * DIM ) );

    // initialize the constant data
    float *temp = (float*)malloc( imageSize );
    for (int i=0; i<DIM*DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x>300) && (x<600) && (y>310) && (y<601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM*100+100] = (MAX_TEMP + MIN_TEMP)/2;
    temp[DIM*700+100] = MIN_TEMP;
    temp[DIM*300+300] = MIN_TEMP;
    temp[DIM*200+700] = MIN_TEMP;
    for (int y=800; y<900; y++) {
        for (int x=400; x<500; x++) {
            temp[x+y*DIM] = MIN_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_constSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );    

    // initialize the input data
    for (int y=800; y<DIM; y++) {
        for (int x=0; x<200; x++) {
            temp[x+y*DIM] = MAX_TEMP;
        }
    }
    HANDLE_ERROR( hipMemcpy( data.dev_inSrc, temp,
                              imageSize,
                              hipMemcpyHostToDevice ) );
    free( temp );

    int ticks=0;
    bitmap_image.show_image(30);
    while(1)
    {
        HANDLE_ERROR( hipEventRecord( data.start, 0 ) );
        dim3    blocks(DIM/16,DIM/16);
        dim3    threads(16,16);
        IMAGE  *bitmap = data.bitmap;

        // since tex is global and bound, we have to use a flag to
        // select which is in/out per iteration
        volatile bool dstOut = true;
        for (int i=0; i<90; i++) {
            float   *in, *out;
            if (dstOut) {
                in  = data.dev_inSrc;
                out = data.dev_outSrc;
            } else {
                out = data.dev_inSrc;
                in  = data.dev_outSrc;
            }
            copy_const_kernel<<<blocks,threads>>>( in );
            blend_kernel<<<blocks,threads>>>( out, dstOut );
            dstOut = !dstOut;
        }
        float_to_color<<<blocks,threads>>>( data.output_bitmap,
                                            data.dev_inSrc );

        HANDLE_ERROR( hipMemcpy( bitmap->get_ptr(),
                                data.output_bitmap,
                                bitmap->image_size(),
                                hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipEventRecord( data.stop, 0 ) );
        HANDLE_ERROR( hipEventSynchronize( data.stop ) );
        float   elapsedTime;
        HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                            data.start, data.stop ) );
        data.totalTime += elapsedTime;
        ++data.frames;
        printf( "Average Time per frame:  %3.1f ms\n",
                data.totalTime/data.frames  );

        ticks++;
        char key = bitmap_image.show_image(30);
        if(key==27)
        {
            break;
        }
    }

    cleanup(&data);
    return 0;
}

