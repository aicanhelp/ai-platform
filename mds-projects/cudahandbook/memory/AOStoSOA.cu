/*
 *
 * AOStoSOA.cu
 *
 * Microdemo that illustrates how to convert from AOS (array of
 * structures) to SOA (structure of arrays) representation.
 *
 * Build with: nvcc -I ../chLib <options> AOStoSOA.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.

 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 

 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include <chError.h>

#include "AOStoSOA_1.cuh"
#include "AOStoSOA_2.cuh"

template<typename T, const int k>
double
TestAOStoSOA( 
    size_t N, 
    void (*pfnAOStoSOA)( T **out, const T *in, size_t N, int cBlocks, int cThreads ),
    int cIterations = 1 )
{
    double ret = 0.0;
    hipError_t status;
    T **dptrpSOA = 0;
    T *dptrSOA[k];
    T *dptrAOS = 0;
    T *hrefSOA[k];  // host reference
    T *hrefAOS = 0;

    hipEvent_t evStart = 0;
    hipEvent_t evStop = 0;

    cuda(EventCreate( &evStart ) );
    cuda(EventCreate( &evStop ) );

    memset( hrefSOA, 0, sizeof(hrefSOA) );
    memset( dptrSOA, 0, sizeof(dptrSOA) );
    cuda(Malloc( &dptrAOS, N*k*sizeof(T) ) );
    cuda(HostAlloc( &hrefAOS, N*k*sizeof(T), hipHostMallocMapped ) );

    cuda(Malloc( &dptrpSOA, k*sizeof(T *) ) );
    for ( int i = 0; i < k; i++ ) {
        cuda(HostAlloc( &hrefSOA[i], N*sizeof(T), hipHostMallocMapped ) );
        memset( hrefSOA[i], 0, N*sizeof(T) );
        cuda(Malloc( &dptrSOA[i], N*sizeof(T) ) );
        cuda(Memset( dptrSOA[i], 0, N*sizeof(T) ) );
    }
    cuda(Memcpy( dptrpSOA, dptrSOA, k*sizeof(T *), hipMemcpyHostToDevice ) );

    for ( size_t i = 0; i < N; i++ ) {
        for ( int j = 0; j < k; j++ ) {
            hrefAOS[i*k+j] = j<<24|i;//rand();
        }
    }
    cuda(MemcpyAsync( dptrAOS, hrefAOS, N*k*sizeof(T), hipMemcpyHostToDevice ) );
    pfnAOStoSOA( dptrpSOA, dptrAOS, N, 1500, 512 );
    for ( int i = 0; i < k; i++ ) {
        cuda(MemcpyAsync( hrefSOA[i], dptrSOA[i], N*sizeof(T), hipMemcpyDeviceToHost ) );
    }
    cuda(DeviceSynchronize() );
    for ( int i = 0; i < N; i++ ) {
        for ( int j = 0; j < k; j++ ) {
            if ( hrefAOS[i*k+j] != hrefSOA[j][i] ) {
                printf( "Mismatch at i==%d, k==%d (%d should be %d)\n", i, j, hrefSOA[j][i], hrefAOS[i*k+j] );
                goto Error;
            }
        }
    }

    cuda(EventRecord( evStart, NULL ) );
    for ( int i = 0; i < cIterations; i++ ) {
        pfnAOStoSOA( dptrpSOA, dptrAOS, N, 1500, 512 );
    }
    cuda(EventRecord( evStop, NULL ) );
    cuda(DeviceSynchronize() );

    {
        float ms;
        cuda(EventElapsedTime( &ms, evStart, evStop ) );
        ret = (double) N*cIterations*sizeof(T)*1000.0 / ms;
    }

Error:
    hipEventDestroy( evStart );
    hipEventDestroy( evStop );
    hipHostFree( hrefAOS );
    hipFree( dptrAOS );
    hipFree( dptrpSOA );
    for ( int i = 0; i < k; i++ ) {
        hipHostFree( hrefSOA[i] );
        hipFree( dptrSOA[i] );
    }
    return ret;
}

int
main( int argc, char *argv[] )
{
    int ret = 1;
    int iN = 32;
    hipError_t status;

    cuda(SetDeviceFlags( hipDeviceMapHost ) );
    cuda(Free(0) );

    #define TEST_VECTOR(fn) { \
        double bytesPerSecond = TestAOStoSOA<int, 3>( 1048576*iN, fn<int,3>, 10 ); \
        if ( 0.0 == bytesPerSecond ) \
            goto Error; \
        printf( "%s: %.2f Gbytes/s\n", #fn, bytesPerSecond/1e9 ); \
    }

    TEST_VECTOR( AOStoSOA_1 );
    TEST_VECTOR( AOStoSOA_2 );

    ret = 0;
Error:
    return ret;
}
