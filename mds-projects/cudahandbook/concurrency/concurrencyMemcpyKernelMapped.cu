/*
 *
 * concurrencyMemcpyKernelMapped.cu
 *
 * Microbenchmark to shmoo the speedup from concurrent kernels when
 * operating on mapped pinned memory.
 *
 * NOTE: To date, I have not found a piece of hardware where this is faster.
 *
 * Build with: nvcc -I ../chLib <options> concurrencyMemcpyKernelMapped.cu
 * Requires: SM 1.1 for mapped pinned memory and global atomics.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.

 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 

 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include "chAssert.h"
#include "chError.h"
#include "chShmoo.h"
#include "chCommandLine.h"
#include "chTimer.h"

#include "AddKernel.cuh"

#include "TimeConcurrentMemcpyKernel.cuh"
#include "TimeSequentialMemcpyKernelMapped.cuh"
#include "TimeConcurrentKernelMapped.cuh"

int
main( int argc, char *argv[] )
{
    const int numTimes = 256;
    float timesSequential[numTimes];
    float timesConcurrent[numTimes];
    int numBlocks;
    int unrollFactor = 1;
    const size_t numInts = 32*1048576;

    hipSetDeviceFlags( hipDeviceMapHost );

    chCommandLineGet( &unrollFactor, "unrollFactor", argc, argv );
    chShmooRange cyclesRange;
    {
        const int minCycles = 8;
        const int maxCycles = 512;
        const int stepCycles = 8;
        cyclesRange.Initialize( minCycles, maxCycles, stepCycles );
        chCommandLineGet( &cyclesRange, "Cycles", argc, argv );
    }

    chShmooRange streamsRange;
    {
        int numStreams = 8;
        if ( ! chCommandLineGet( &streamsRange, "streams", argc, argv ) ) {
            streamsRange.Initialize( numStreams );
        }
    }
        
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties( &props, 0 );
        int multiplier = 16;
        chCommandLineGet( &multiplier, "blocksPerSM", argc, argv );
        numBlocks = props.multiProcessorCount * multiplier;
        printf( "Using %d blocks per SM on GPU with %d SMs = %d blocks\n", multiplier, 
            props.multiProcessorCount, numBlocks );
    }

    printf( "Timing mapped operations" );
    if ( ! TimeSequentialMemcpyKernelMapped( timesSequential, numInts, cyclesRange, numBlocks, unrollFactor ) ) {
        printf( "TimeSequentialMemcpyKernelMapped failed\n" );
        return 1;
    }
    printf( "\nTiming streamed operations" );
    if ( ! TimeConcurrentKernelMapped( timesConcurrent, numInts, cyclesRange, streamsRange, numBlocks, unrollFactor ) ) {
        printf( "TimeConcurrentMemcpyKernel failed\n" );
        return 1;
    }

    printf( "\n%d integers\n", (int) numInts );
    printf( "Cycles\tMapped\tStreamed\tSpeedup\n" );

    int index = 0;
    for ( chShmooIterator cycles(cyclesRange); cycles; cycles++, index++ ) {
        printf( "%d\t%.2f\t%.2f\t%.2f\n", 
            *cycles, timesSequential[index], timesConcurrent[index],
            timesConcurrent[index] / timesSequential[index] );
    }

    return 0;
}
