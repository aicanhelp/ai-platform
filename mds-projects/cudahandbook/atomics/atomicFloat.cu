
#include <hip/hip_runtime.h>
/*
 *
 * atomicFloat.cu
 *
 * Microdemo for atomic floating point addition in global memory.
 *
 * NOT INTENDED AS A SAMPLE FOR ANYTHING OTHER THAN CODE GENERATION.
 *
 * Build with: 
 *     nvcc --gpu-architecture sm_20 --cubin atomicFloat.cu
 *     cuobjdump --dump-sass atomicFloat.cubin
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

__global__ void
AddToFloat( float *sum, float *out, const float *pIn )
{
    (void) atomicAdd( &out[threadIdx.x], pIn[threadIdx.x] );
}

__global__ void
ReturnFloat( float *sum, float *out, const float *pIn )
{
    out[threadIdx.x] = atomicAdd( &out[threadIdx.x], pIn[threadIdx.x] );
}
