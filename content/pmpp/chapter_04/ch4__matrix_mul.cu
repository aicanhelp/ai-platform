#include "hip/hip_runtime.h"
/*
 * chapter_4.cu
 *
 *  Created on: 27/11/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <string.h>
#include <math.h>
#include "ch4__config.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__array_utils.h"
#include "nvixnu__error_utils.h"
#include "pmpp__blas.h"


void ch4__matrix_mul_device(double *h_A, double *h_B, double *h_C, const int i_length, const int j_length, const int k_length, kernel_config_t config){
	double *d_A, *d_B, *d_C;

	const int A_LENGTH = i_length*j_length;
	const int B_LENGTH = j_length*k_length;
	const int C_LENGTH = i_length*k_length;

	CCE(hipMalloc(&d_A, A_LENGTH*sizeof(double)));
	CCE(hipMalloc(&d_B, B_LENGTH*sizeof(double)));
	CCE(hipMalloc(&d_C, C_LENGTH*sizeof(double)));


	CCE(hipMemcpy(d_A, h_A, A_LENGTH*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_B, h_B, B_LENGTH*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_C, h_C, C_LENGTH*sizeof(double), hipMemcpyHostToDevice));

	dim3 block_dim(config.block_dim.x, config.block_dim.y, 1);
	dim3 grid_dim(ceil(k_length/(double)config.block_dim.x), ceil(i_length/(double)config.block_dim.y), 1);

	DEVICE_TIC(0);
	if(!strcmp(config.kernel_version, CH4__MATRIX_MUL_KERNEL_NAIVE)){
		pmpp__gemm_kernel<<<grid_dim, block_dim>>>(d_A, d_B, d_C, i_length, j_length, k_length);
	}else if(!strcmp(config.kernel_version, CH4__MATRIX_MUL_KERNEL_TILED)){
		const int shared_memory_length = 2*config.block_dim.x*config.block_dim.y*sizeof(double);
		pmpp__tiled_gemm_kernel<<<grid_dim, block_dim, shared_memory_length>>>(d_A, d_B, d_C, i_length, j_length, k_length, config.block_dim.x);
	}else{
		printf("\nINVALID KERNEL VERSION\n");
		exit(1);
	}
	CCLE();
	DEVICE_TOC(0);

	CCE(hipMemcpy(h_C, d_C, C_LENGTH*sizeof(double), hipMemcpyDeviceToHost));


	CCE(hipFree(d_A));
	CCE(hipFree(d_B));
	CCE(hipFree(d_C));
}

void ch4__matrix_mul_host(double *A, double *B, double *C, const int i_length, const int j_length, const int k_length){
	HOST_TIC(0);
	pmpp__gemm_host(A, B, C, i_length, j_length, k_length);
	HOST_TOC(0);
}

void ch4__matrix_mul(env_e env, kernel_config_t config){
	double *A, *B, *C;

	const int A_LENGTH = CH4__I_LENGTH*CH4__J_LENGTH;
	const int B_LENGTH = CH4__J_LENGTH*CH4__K_LENGTH;
	const int C_LENGTH = CH4__I_LENGTH*CH4__K_LENGTH;

	A = (double*)malloc(A_LENGTH*sizeof(double));
	B = (double*)malloc(B_LENGTH*sizeof(double));
	C = (double*)calloc(C_LENGTH, sizeof(double));

	nvixnu__populate_array_from_file(CH4__MATRIX_A_PATH, "%lf,", A_LENGTH, sizeof(double), A);
	nvixnu__populate_array_from_file(CH4__MATRIX_B_PATH, "%lf,", B_LENGTH, sizeof(double), B);

	if(env == Host){
		ch4__matrix_mul_host(A, B, C, CH4__I_LENGTH, CH4__J_LENGTH, CH4__K_LENGTH);
	}else{
		ch4__matrix_mul_device(A, B, C, CH4__I_LENGTH, CH4__J_LENGTH, CH4__K_LENGTH, config);
	}


	printf("Last %d values:\n", PRINT_LENGTH);
	nvixnu__array_map(C + (C_LENGTH - PRINT_LENGTH), sizeof(double), PRINT_LENGTH, nvixnu__print_item_double);

	free(A);
	free(B);
	free(C);

	return;
}


int main(){
	printf("Chapter 04\n");
	printf("Matrix A: %dx%d\n", CH4__I_LENGTH, CH4__J_LENGTH);
	printf("Matrix B: %dx%d\n", CH4__J_LENGTH, CH4__K_LENGTH);

	printf("\n_____ matrix_mul _____\n\n");

	printf("Running on Device with 256 threads per block...");
	ch4__matrix_mul(Device, {.block_dim = {16,16,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_NAIVE});

	printf("\nRunning  on Device with 1024 threads per block...");
	ch4__matrix_mul(Device, {.block_dim = {32,32,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_NAIVE});

	printf("\n_____ matrix_mul_tiled _____\n");

	printf("\nRunning  on Device with 256 threads per block...");
	ch4__matrix_mul(Device, {.block_dim = {16,16,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_TILED});

	printf("\nRunning on Device with 1024 threads per block...");
	ch4__matrix_mul(Device, {.block_dim = {32,32,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_TILED});

	printf("\n_____ matrix_mul_CPU _____\n");
	ch4__matrix_mul(Host, {});

	return 0;
}

