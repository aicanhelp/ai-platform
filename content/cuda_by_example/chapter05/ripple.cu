#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "book.h"
#include "image.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf( fx * fx + fy * fy );
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                                         cos(d/10.0f - ticks/7.0f) /
                                         (d/10.0f + 1.0f));    
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255;
}

struct DataBlock {
    unsigned char   *dev_bitmap;
    IMAGE  *bitmap;
};


// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

int main( void ) {
    DataBlock   data;
    IMAGE  bitmap( DIM, DIM );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );

    dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
    
    int ticks = 0;
    bitmap.show_image(30);
    while(1)
    {
        kernel<<<blocks,threads>>>( data.dev_bitmap, ticks );

        HANDLE_ERROR( hipMemcpy( data.bitmap->get_ptr(),
                            data.dev_bitmap,
                            data.bitmap->image_size(),
                            hipMemcpyDeviceToHost ) );

        ticks++;
        char key = bitmap.show_image(30);
        if(key==27)
        {
            break;
        }
    }

	cleanup(&data);

	return 0;
}
