#include "hip/hip_runtime.h"
/*
 *
 * peer2peerMemcpyDrv.cu
 *
 * Driver API version of the sample shows how to use portable 
 * pinned memory and inter-GPU synchronization to perform a 
 * peer-to-peer memcpy.
 *
 * Build with: nvcc -I ../chLib <options> peer2peerMemcpyDrv.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */


#include <stdio.h>

#include <hip/hip_runtime.h>

#include "chError.h"
#include "chTimer.h"

#define STAGING_BUFFER_SIZE 1048576

void *g_hostBuffers[2];

// Indexed as follows: [device][event]
hipEvent_t g_events[2][2];

// these are already defined on some platforms - make our
// own definitions that will work.
#undef min
#undef max
#define min(a,b) ((a)<(b)?(a):(b))
#define max(a,b) ((b)<(a)?(a):(b))

hipError_t
chMemcpyPeerToPeer( 
    void *_dst, hipCtx_t dstContext, int dstDevice,
    const void *_src, hipCtx_t srcContext, int srcDevice,
    size_t N ) 
{
    hipError_t status;
    hipDeviceptr_t dst = (hipDeviceptr_t) (intptr_t) _dst;
    hipDeviceptr_t src = (hipDeviceptr_t) (intptr_t) _src;
    int stagingIndex = 0;

    while ( N ) {
        size_t thisCopySize = min( N, STAGING_BUFFER_SIZE );

        cu(CtxPushCurrent( srcContext ) );
        cu(StreamWaitEvent( 
            NULL, g_events[dstDevice][stagingIndex], 0 ) );
        cu(MemcpyDtoHAsync( 
            g_hostBuffers[stagingIndex], 
            src, 
            thisCopySize, 
            NULL ) );
        cu(EventRecord( 
            g_events[srcDevice][stagingIndex], 
            0 ) );

        cu(CtxPopCurrent( &srcContext ) );
        cu(CtxPushCurrent( dstContext ) );
        cu(StreamWaitEvent( 
            NULL, 
            g_events[srcDevice][stagingIndex], 
            0 ) );
        cu(MemcpyHtoDAsync( 
            dst, 
            g_hostBuffers[stagingIndex], 
            thisCopySize, 
            NULL ) );
        cu(EventRecord( 
            g_events[dstDevice][stagingIndex], 
            0 ) );

        cu(CtxPopCurrent( &dstContext ) );

        dst += thisCopySize;
        src += thisCopySize;
        N -= thisCopySize;
        stagingIndex = 1 - stagingIndex;
    }

    // Wait until both devices are done
    cu(CtxPushCurrent( srcContext ) );
    cu(CtxSynchronize() );
    cu(CtxPopCurrent( &srcContext ) );

    cu(CtxPushCurrent( dstContext ) );
    cu(CtxSynchronize() );
    cu(CtxPopCurrent( &dstContext ) );
    
Error:
    return status;
}

bool
TestMemcpy( 
    int *dst, int dstDevice,
    int *src, int srcDevice,
    int *srcHost, const int *srcOriginal,
    size_t dstOffset, size_t srcOffset, 
    size_t numInts )
{
    hipError_t status;
    hipCtx_t srcContext, dstContext;

    memset( srcHost, 0, numInts );
    hipSetDevice( dstDevice );
    if ( hipSuccess != hipCtxGetCurrent( &dstContext ) )
        return false;
    hipSetDevice( srcDevice );
    if ( hipSuccess != hipCtxGetCurrent( &srcContext ) )
        return false;

    cuda(Memcpy( src+srcOffset, srcOriginal+srcOffset, 
        numInts*sizeof(int), hipMemcpyHostToDevice ) );
    memset( srcHost, 0, numInts*sizeof(int) );
    chMemcpyPeerToPeer( dst+dstOffset, dstContext, dstDevice, 
                        src+srcOffset, srcContext, srcDevice, 
                        numInts*sizeof(int) );
    cuda(Memcpy( srcHost, dst+dstOffset, numInts*sizeof(int), hipMemcpyDeviceToHost ) );
    for ( size_t i = 0; i < numInts; i++ ) {
        if ( srcHost[i] != srcOriginal[srcOffset+i] ) {
            return false;
        }
    }
    return true;
Error:
    return false;
}

int
main( int argc, char *argv[] )
{
    int deviceCount;

    hipError_t status;
    int *deviceInt[2];
    int *hostInt = 0;
    const size_t numInts = 8*1048576;
    const int cIterations = 10;
    int *testVector = 0;

    hipCtx_t srcContext, dstContext;


    printf( "Peer-to-peer memcpy... " ); fflush( stdout );

    chTimerTimestamp start, stop;

    memset( deviceInt, 0, sizeof(deviceInt) );

    cuda(GetDeviceCount( &deviceCount ) );

    if ( deviceCount <= 1 ) {
        printf( "Peer-to-peer demo requires at least 2 devices\n" );
        exit(1);
    }

    for ( int i = 0; i < 2; i++ ) {
        hipSetDevice( i );

        cuda(EventCreate( &g_events[i][0] ) );
        cuda(EventRecord( g_events[i][0], 0 ) );  // so it is signaled on first synchronize
        cuda(EventCreate( &g_events[i][1] ) );
        cuda(EventRecord( g_events[i][1], 0 ) );  // so it is signaled on first synchronize

        cuda(Malloc( &deviceInt[i], numInts*sizeof(int) ) );
    }

    cuda(HostAlloc( &g_hostBuffers[0], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );
    cuda(HostAlloc( &g_hostBuffers[1], STAGING_BUFFER_SIZE, hipHostMallocPortable ) );

    cuda(HostAlloc( &hostInt, numInts*sizeof(int), 0 ) );

    testVector = (int *) malloc( numInts*sizeof(int) );
    if ( ! testVector ) {
        printf( "malloc() failed\n" );
        return 1;
    }
    for ( size_t i = 0; i < numInts; i++ ) {
        testVector[i] = rand();
    }

    cuda(SetDevice( 0 ) );
    if ( hipSuccess != hipCtxGetCurrent( &dstContext ) )
        goto Error;
    cuda(SetDevice( 1 ) );
    if ( hipSuccess != hipCtxGetCurrent( &srcContext ) )
        goto Error;
    if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, 
                       hostInt, testVector, 0, 0, numInts ) ) {
        goto Error;
    }

    for ( int i = 0; i < cIterations; i++ ) {
        size_t dstOffset = rand() % (numInts-1);
        size_t srcOffset = rand() % (numInts-1);
        size_t intsThisIteration = 1 + rand() % (numInts-max(dstOffset,srcOffset)-1);
        if ( ! TestMemcpy( deviceInt[0], 0, deviceInt[1], 1, hostInt, testVector, dstOffset, srcOffset, intsThisIteration ) ) {
            //TestMemcpy( deviceInt, hostInt, testVector, dstOffset, srcOffset, intsThisIteration );
            goto Error;
        }
    }

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        chMemcpyPeerToPeer( deviceInt[0], dstContext, 0, deviceInt[1], srcContext, 1, numInts*sizeof(int) ) ;
    }
    cuda(DeviceSynchronize() );
    chTimerGetTime( &stop );

    {
        double MBytes = cIterations*numInts*sizeof(int) / 1048576.0;
        double MBpers = MBytes / chTimerElapsedTime( &start, &stop );

        printf( "%.2f MB/s\n", MBpers );
    }

    hipFree( deviceInt );
    hipHostFree( hostInt );
    return 0;
Error:
    printf( "Error\n" );
    return 1;
}
