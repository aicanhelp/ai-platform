#include "hip/hip_runtime.h"
/*
 *
 * win32KernelStreamCallback.cu
 *
 * Microdemo to examine the behavior of stream callbacks on Win32.
 *
 * Periodically counts the number of CPU threads extant, and reports
 * that number.
 *
 * Build with: nvcc -I ../chLib <options> win32KernelStreamCallback.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>
#include <chTimer.h>

#include <tlhelp32.h>
#include <tchar.h>

__global__
void
NullKernel()
{
}

__global__
void
DereferenceNullKernel()
{
    *((volatile int *) 0) = 0xdeadbeef;
}

class CStreamCallbacksStats {
private:
    CRITICAL_SECTION m_cs;

    int m_cCallbacks;
    int m_intervalCount;
    int m_totalCallbacks;

    DWORD m_dwThreadId;

    HANDLE m_hEvent;      // event to signal
    int m_signalCount;    // number of callbacks to look for before signaling event
    int m_maxThreadCount;

    static const int intervalPeriod = 1000;


public:
    CStreamCallbacksStats() {
        InitializeCriticalSection( &m_cs );
        m_hEvent = NULL;
        m_signalCount = 0;
        m_intervalCount = intervalPeriod;
        m_cCallbacks = 0;
        m_totalCallbacks = 0;
        m_maxThreadCount = 0;
        m_dwThreadId = 0;
    }
    ~CStreamCallbacksStats() {
        DeleteCriticalSection( &m_cs );
        if ( m_hEvent ) {
            CloseHandle( m_hEvent );
        }
    }
    bool Initialize( int numCallbacksBeforeSignal ) {
        m_signalCount = numCallbacksBeforeSignal;
        m_hEvent = CreateEvent( NULL, FALSE, FALSE, NULL );
        return m_hEvent != NULL;
    }

    int getTotalCallbacks() const { return m_totalCallbacks; }

    static void CUDART_CB countCallbacks( hipStream_t stream, hipError_t status, void *userData );
    BOOL Wait() { return WaitForSingleObject( m_hEvent, INFINITE ); }

    int CountProcessThreads( );
} ;

int
CStreamCallbacksStats::CountProcessThreads( ) 
{
    DWORD dwOwnerPID = GetCurrentProcessId();
    int ret = 0;
    int numThreads = 0;
    HANDLE hThreadSnap = INVALID_HANDLE_VALUE; 
    THREADENTRY32 te32; 
 
    // Take a snapshot of all running threads  
    hThreadSnap = CreateToolhelp32Snapshot( TH32CS_SNAPTHREAD, 0 ); 
    if( hThreadSnap == INVALID_HANDLE_VALUE ) 
        return( FALSE ); 

    // Fill in the size of the structure before using it. 
    te32.dwSize = sizeof(THREADENTRY32 ); 
 
    // Retrieve information about the first thread,
    // and exit if unsuccessful
    if( !Thread32First( hThreadSnap, &te32 ) ) 
        goto Error;
    numThreads = 0;
    do {
        if( te32.th32OwnerProcessID == dwOwnerPID ) {
            numThreads += 1;
        }
    } while( Thread32Next(hThreadSnap, &te32 ) );

    EnterCriticalSection( &m_cs );
    if ( numThreads > m_maxThreadCount ) {
        m_maxThreadCount = numThreads;
    }
    LeaveCriticalSection( &m_cs );
    ret = numThreads;
Error:
    CloseHandle( hThreadSnap );
    return ret;
}


void CUDART_CB 
CStreamCallbacksStats::countCallbacks( hipStream_t stream, hipError_t status, void *userData )
{
    CStreamCallbacksStats *p = (CStreamCallbacksStats *) userData;
    const int intervalPeriod = 1000;

    EnterCriticalSection( &p->m_cs );
        DWORD dwThreadId = GetCurrentThreadId();
        if ( p->m_dwThreadId == 0 ) {
            printf( "Initializing thread ID\n" );
            p->m_dwThreadId = dwThreadId;
        }
        else {
            if ( p->m_dwThreadId != dwThreadId ) {
                printf( "Different thread ID\n" );
            }
        }
    LeaveCriticalSection( &p->m_cs );

    if ( 0 == InterlockedDecrement( (LPLONG) &p->m_intervalCount ) ) {
        InterlockedExchange( (LPLONG) &p->m_intervalCount, intervalPeriod );
        p->CountProcessThreads( );
    }

    if ( hipSuccess != status ) {
        // confirm that kernel that faulted is reported properly
        printf( "status = %d\n", status );
        return;
    }
    if ( p->m_signalCount == InterlockedIncrement( (LONG *) &p->m_totalCallbacks ) ) {
        printf( "Signaling event\n" );
        SetEvent( p->m_hEvent );
    }
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    const int cIterations = 1000;

    CStreamCallbacksStats stats;

    stats.Initialize( cIterations );

    hipInit(0);//
    Sleep(1000);
    printf( "Max threads after hipInit(0): %d\n", stats.CountProcessThreads() );
    hipFree(0);
    printf( "Max threads after hipFree(0): %d\n", stats.CountProcessThreads() );
cuda(StreamAddCallback( NULL, CStreamCallbacksStats::countCallbacks, &stats, cudaStreamCallbackNonblocking ) );

    printf( "Max threads after hipStreamAddCallback(): %d\n", stats.CountProcessThreads() );

    printf( "Measuring asynchronous launch time (with nonblocking callbacks)... " ); fflush( stdout );

    chTimerTimestamp start, stop;

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
        NullKernel<<<1,1>>>();
        cuda(StreamAddCallback( NULL, CStreamCallbacksStats::countCallbacks, &stats, cudaStreamCallbackNonblocking ) );
        if ( i == 0 ) {
            printf( "Max threads: %d\n", stats.CountProcessThreads() );
        }
    }
    DereferenceNullKernel<<<1,1>>>();
    cuda(StreamAddCallback( NULL, CStreamCallbacksStats::countCallbacks, &stats, cudaStreamCallbackNonblocking ) );
    hipDeviceSynchronize();
    chTimerGetTime( &stop );

    // race condition unless we wait here
    stats.Wait();

    printf( "%d callbacks\n", stats.getTotalCallbacks() );
    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f us\n", usPerLaunch );
    }

    return 0;
Error:
    printf( "CUDA error: %d (%s)\n", status, hipGetErrorString( status ) );
    return 1;
}
