
#include <hip/hip_runtime.h>
/*
 *
 * atomic64.cu
 *
 * Microdemo for atomic operations on 64-bit integers in global memory.
 *
 * NOT INTENDED AS A SAMPLE FOR ANYTHING OTHER THAN CODE GENERATION.
 *
 * Build with: 
 *     nvcc --gpu-architecture sm_xx --cubin atomic64.cu
 *     cuobjdump --dump-sass atomic64.cubin
 * (fill in xx with 11, 20, or 30 to examine code generation of Tesla,
 * Fermi and Kepler, respectively)
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

__global__ void
Return64( unsigned long long *sum, unsigned long long *out, const unsigned long long *pIn )
{
    out[threadIdx.x] = atomicAdd( &sum[threadIdx.x], *pIn );
}
