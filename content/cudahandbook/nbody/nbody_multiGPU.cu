#include "hip/hip_runtime.h"
/*
 *
 * nbody_multiGPU.cu
 *
 * Single-threaded multi-GPU implementation of the O(N^2) N-body calculation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include <chError.h>
#include <chTimer.h>
#include <chThread.h>

#include "nbody.h"
#include "nbody_multiGPU_shared.cuh"

#include "bodybodyInteraction.cuh"

using namespace cudahandbook::threading;

__global__ void
ComputeNBodyGravitation_multiGPU_onethread( 
    float *force, 
    float *posMass, 
    float softeningSquared, 
    size_t base, 
    size_t n, 
    size_t N )
{
    ComputeNBodyGravitation_Shared_multiGPU( 
        force, 
        posMass, 
        softeningSquared, 
        base, 
        n,
        N );
}

float
ComputeGravitation_multiGPU_singlethread( 
    float *force, 
    float *posMass,
    float softeningSquared,
    size_t N
)
{
    hipError_t status;

    float ret = 0.0f;

    float *dptrPosMass[g_maxGPUs];
    float *dptrForce[g_maxGPUs];
    int oldDevice;

    chTimerTimestamp start, end;
    chTimerGetTime( &start );

    memset( dptrPosMass, 0, sizeof(dptrPosMass) );
    memset( dptrForce, 0, sizeof(dptrForce) );
    size_t bodiesPerGPU = N / g_numGPUs;
    if ( (0 != N % g_numGPUs) || (g_numGPUs > g_maxGPUs) ) {
        return 0.0f;
    }
    cuda(GetDevice( &oldDevice ) );

    // kick off the asynchronous memcpy's - overlap GPUs pulling
    // host memory with the CPU time needed to do the memory 
    // allocations.
    for ( int i = 0; i < g_numGPUs; i++ ) {
        cuda(SetDevice( i ) );
        cuda(Malloc( &dptrPosMass[i], 4*N*sizeof(float) ) );
        // we only need 3*N floatsw for the cross-check. otherwise we 
        // would need 3*bodiesPerGPU
        cuda(Malloc( &dptrForce[i], 3*N*sizeof(float) ) );
        cuda(MemcpyAsync( 
            dptrPosMass[i], 
            g_hostAOS_PosMass, 
            4*N*sizeof(float), 
            hipMemcpyHostToDevice ) );
    }
    for ( int i = 0; i < g_numGPUs; i++ ) {
        cuda(SetDevice( i ) );
        if ( g_bGPUCrossCheck ) {
            ComputeNBodyGravitation_multiGPU_onethread<<<300,256,256*sizeof(float4)>>>( 
                dptrForce[i],
                dptrPosMass[i],
                softeningSquared,
                0,
                N,
                N );
            cuda(MemcpyAsync( 
                g_hostAOS_gpuCrossCheckForce[i], 
                dptrForce[i], 
                3*N*sizeof(float), 
                hipMemcpyDeviceToHost ) );
            cuda(MemcpyAsync( 
                g_hostAOS_Force+3*bodiesPerGPU*i, 
                dptrForce[i]+3*bodiesPerGPU*i, 
                3*bodiesPerGPU*sizeof(float), 
                hipMemcpyDeviceToHost ) );
        }
        else {
            ComputeNBodyGravitation_multiGPU_onethread<<<300,256,256*sizeof(float4)>>>( 
                dptrForce[i],
                dptrPosMass[i],
                softeningSquared,
                i*bodiesPerGPU,
                bodiesPerGPU,
                N );
            cuda(MemcpyAsync( 
                g_hostAOS_Force+3*bodiesPerGPU*i, 
                dptrForce[i], 
                3*bodiesPerGPU*sizeof(float), 
                hipMemcpyDeviceToHost ) );
        }
    }
    // Synchronize with each GPU in turn.
    for ( int i = 0; i < g_numGPUs; i++ ) {
        cuda(SetDevice( i ) );
        cuda(DeviceSynchronize() );
    }
    chTimerGetTime( &end );
    ret = chTimerElapsedTime( &start, &end ) * 1000.0f;

    if ( g_fGPUCrosscheckOutput ) {
        if ( 1 != fwrite( g_hostAOS_Force, 3*N*sizeof(float), 1, g_fGPUCrosscheckOutput ) )
            goto Error;
    }
    if ( g_fGPUCrosscheckInput ) {
        if ( 1 != fread( g_hostAOS_Force_Golden, 3*N*sizeof(float), 1, g_fGPUCrosscheckInput ) )
            goto Error;
    }


Error:
    for ( int i = 0; i < g_numGPUs; i++ ) {
        hipFree( dptrPosMass[i] );
        hipFree( dptrForce[i] );
    }
    hipSetDevice( oldDevice );
    return ret;
}
